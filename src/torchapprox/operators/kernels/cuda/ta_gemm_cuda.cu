#include "hip/hip_runtime.h"
#include "ta_gemm_cuda.h"

const auto BLOCK_SIZE = 16;
const auto K = 5;

__device__ inline int32_t DRUM(int16_t op1, int16_t op2) {
    if (op1 == 0 || op2 == 0)
        return 0;
    if (op1 == -1)
        return -op2;
    if (op2 == -1)
        return -op1;

    // Sign extraction
    const bool sgn1 = op1 < 0;
    const bool sgn2 = op2 < 0;

    uint32_t abs1 = sgn1 ? -op1 : op1;
    uint32_t abs2 = sgn2 ? -op2 : op2;

    // Find leading one
    const auto lead1_1 = 31 - __clz(abs1);
    const auto lead1_2 = 31 - __clz(abs2);

    // Mask with the lowest `k` Bits set, zero otherwise
    const auto mask = (1 << K) - 1;
    if (lead1_1 > K) {
        // Truncate to the most-significant `k` bits
        abs1 &= (mask << (lead1_1 - K + 1));
        // Always set lowest non-truncated Bit position to 1
        abs1 |= (1 << (lead1_1 - K + 1));
    }
    if (lead1_2 > K) {
        abs2 &= (mask << (lead1_2 - K + 1));
        abs2 |= (1 << (lead1_2 - K + 1));
    }

    // This derives from the hardware implementation in that
    // we perform a regular multiplication instead of
    // adding and shifting to keep things simple.
    // The result is the same, however, because the approximation
    // has already been applied to the operands at this point.
    auto y0 = abs1 * abs2;
    auto y = (sgn1 ^ sgn2) ? -y0 : y0;

    return y;
}

__device__ inline int32_t mitchell_trunc(int16_t op1, int16_t op2) {
    // Same as DRUM, only that the lowest non-truncated Bit position is not
    // de-biased by setting it to one.
    if (op1 == 0 || op2 == 0)
        return 0;
    if (op1 == -1)
        return -op2;
    if (op2 == -1)
        return -op1;

    // Sign extraction
    const bool sgn1 = op1 < 0;
    const bool sgn2 = op2 < 0;

    uint32_t abs1 = sgn1 ? -op1 : op1;
    uint32_t abs2 = sgn2 ? -op2 : op2;

    // Find leading one
    const auto lead1_1 = 31 - __clz(abs1);
    const auto lead1_2 = 31 - __clz(abs2);

    // Mask with the lowest `k` Bits set, zero otherwise
    const auto mask = (1 << K) - 1;
    if (lead1_1 > K) {
        // Truncate to the most-significant `k` bits
        abs1 &= (mask << (lead1_1 - K + 1));
    }
    if (lead1_2 > K) {
        abs2 &= (mask << (lead1_2 - K + 1));
    }

    auto y0 = abs1 * abs2;
    auto y = (sgn1 ^ sgn2) ? -y0 : y0;

    return y;
}

template <typename scalar_t>
__device__ inline int32_t lut_operator(hipTextureObject_t tex, scalar_t idx1, scalar_t idx2) {
    auto i1 = static_cast<uint8_t>(idx1);
    auto i2 = static_cast<uint8_t>(idx2);
    auto idx = (i1 << 8) | i2;
    return tex1Dfetch<int32_t>(tex, idx);
}

template <typename scalar_t>
__global__ void
ta_gemm_kernel(hipTextureObject_t tex,
               const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> a,
               const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> b,
               torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> res) {
    __shared__ scalar_t a_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ scalar_t b_shared[BLOCK_SIZE][BLOCK_SIZE];

    const auto col = blockIdx.x * blockDim.x + threadIdx.x;
    const auto row = blockIdx.y * blockDim.y + threadIdx.y;
    const auto batch = blockIdx.z * blockDim.z + threadIdx.z;

    int32_t acc = 0;

    for (auto tile_offset = 0; tile_offset < (a.size(2) + BLOCK_SIZE - 1) / BLOCK_SIZE;
         tile_offset++) {
        if (tile_offset * BLOCK_SIZE + threadIdx.x < a.size(2) && row < a.size(1)) {
            a_shared[threadIdx.y][threadIdx.x] =
                a[batch][row][tile_offset * BLOCK_SIZE + threadIdx.x];
        } else {
            a_shared[threadIdx.y][threadIdx.x] = 0;
        }

        if (tile_offset * BLOCK_SIZE + threadIdx.y < b.size(1) && col < b.size(0)) {
            b_shared[threadIdx.x][threadIdx.y] = b[col][tile_offset * BLOCK_SIZE + threadIdx.y];
        } else {
            b_shared[threadIdx.x][threadIdx.y] = 0;
        }

        __syncthreads();

#pragma unroll
        for (auto n = 0; n < BLOCK_SIZE; n++) {
            auto i1 = a_shared[threadIdx.y][n];
            auto i2 = b_shared[threadIdx.x][n];
            /* auto val = lut_operator<uint8_t>(tex, i1, i2);*/
            auto val = mitchell_trunc((int16_t)i1, (int16_t)i2);
            acc += val;
        }
        __syncthreads();
    }

    if (batch < res.size(0) && row < res.size(1) && col < res.size(2)) {
        res[batch][row][col] = acc;
    }
}

template <typename scalar_t>
__global__ void
ta_gemm_kernel_batchb(hipTextureObject_t tex,
                      const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> a,
                      const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> b,
                      torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> res) {
    __shared__ scalar_t a_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ scalar_t b_shared[BLOCK_SIZE][BLOCK_SIZE];

    const auto col = blockIdx.x * blockDim.x + threadIdx.x;
    const auto row = blockIdx.y * blockDim.y + threadIdx.y;
    const auto batch = blockIdx.z * blockDim.z + threadIdx.z;

    int32_t acc = 0;

    for (auto tile_offset = 0; tile_offset < (a.size(1) + BLOCK_SIZE - 1) / BLOCK_SIZE;
         tile_offset++) {
        if (tile_offset * BLOCK_SIZE + threadIdx.x < a.size(1) && row < a.size(0)) {
            a_shared[threadIdx.y][threadIdx.x] = a[row][tile_offset * BLOCK_SIZE + threadIdx.x];
        } else {
            a_shared[threadIdx.y][threadIdx.x] = 0;
        }

        if (tile_offset * BLOCK_SIZE + threadIdx.y < b.size(2) && col < b.size(1)) {
            b_shared[threadIdx.x][threadIdx.y] =
                b[batch][col][tile_offset * BLOCK_SIZE + threadIdx.y];
        } else {
            b_shared[threadIdx.x][threadIdx.y] = 0;
        }

        __syncthreads();

#pragma unroll
        for (auto n = 0; n < BLOCK_SIZE; n++) {
            auto i1 = a_shared[threadIdx.y][n];
            auto i2 = b_shared[threadIdx.x][n];
            /* auto val = lut_operator<uint8_t>(tex, i2, i1);*/
            auto val = mitchell_trunc((int16_t)i2, (int16_t)i1);
            acc += val;
        }
        __syncthreads();
    }

    if (batch < res.size(0) && row < res.size(1) && col < res.size(2)) {
        res[batch][row][col] = acc;
    }
}

void ta_gemm_cuda_launch(at::Tensor a, at::Tensor b, at::Tensor lut, at::Tensor res) {
    // prepare the kernel configuration
    const dim3 blocks((res.size(2) + BLOCK_SIZE - 1) / BLOCK_SIZE,
                      (res.size(1) + BLOCK_SIZE - 1) / BLOCK_SIZE, res.size(0));
    const dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Create resource description
    struct hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = lut.data_ptr<int32_t>();
    resDesc.res.linear.sizeInBytes = lut.size(0) * lut.size(1) * sizeof(int32_t);
    resDesc.res.linear.desc = hipCreateChannelDesc<int32_t>();

    // Create texture description
    struct hipTextureDesc texDesc = {};
    texDesc.readMode = hipReadModeElementType;

    // Create texture
    hipTextureObject_t tex;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

    if (a.dim() == 3) {
        AT_DISPATCH_ALL_TYPES(
            a.scalar_type(), "torchapprox cuda", ([&] {
                ta_gemm_kernel<scalar_t><<<blocks, threads_per_block>>>(
                    tex, a.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                    b.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    res.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>());
            }));

    } else {
        AT_DISPATCH_ALL_TYPES(
            a.scalar_type(), "torchapprox cuda", ([&] {
                ta_gemm_kernel_batchb<scalar_t><<<blocks, threads_per_block>>>(
                    tex, a.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    b.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                    res.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>());
            }));
    }
}
